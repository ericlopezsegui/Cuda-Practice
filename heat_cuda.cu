#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define ALPHA 0.01
#define DX 0.02
#define DY 0.02
#define DT 0.0005
#define T 1500.0
#define BMP_HEADER_SIZE 54

__device__ double compute_point(double *grid, int i, int j, int nx, int ny, double r) {
    int idx = i * ny + j;
    return grid[idx]
           + r * (grid[(i + 1) * ny + j] + grid[(i - 1) * ny + j] - 2 * grid[idx])
           + r * (grid[i * ny + j + 1] + grid[i * ny + j - 1] - 2 * grid[idx]);
}

__global__ void heat_step(double *grid, double *new_grid, int nx, int ny, double r) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1) {
        int idx = i * ny + j;
        new_grid[idx] = grid[idx]
            + r * (grid[(i + 1) * ny + j] + grid[(i - 1) * ny + j] - 2 * grid[idx])
            + r * (grid[i * ny + j + 1] + grid[i * ny + j - 1] - 2 * grid[idx]);
    }
}

void initialize_grid(double *grid, int nx, int ny) {
    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            if (i == j || i == nx - 1 - j)
                grid[i * ny + j] = T;
            else
                grid[i * ny + j] = 0.0;
        }
    }

    for (int i = 0; i < nx; i++) {
        grid[i * ny + 0] = 0.0;
        grid[i * ny + (ny - 1)] = 0.0;
    }
    for (int j = 0; j < ny; j++) {
        grid[0 * ny + j] = 0.0;
        grid[(nx - 1) * ny + j] = 0.0;
    }
}

void write_bmp_header(FILE *file, int width, int height) {
    unsigned char header[BMP_HEADER_SIZE] = {0};
    int file_size = BMP_HEADER_SIZE + 3 * width * height;
    header[0] = 'B'; header[1] = 'M';
    header[2] = file_size & 0xFF; header[3] = (file_size >> 8) & 0xFF;
    header[4] = (file_size >> 16) & 0xFF; header[5] = (file_size >> 24) & 0xFF;
    header[10] = BMP_HEADER_SIZE;
    header[14] = 40;
    header[18] = width & 0xFF; header[19] = (width >> 8) & 0xFF;
    header[22] = height & 0xFF; header[23] = (height >> 8) & 0xFF;
    header[26] = 1; header[28] = 24;
    fwrite(header, 1, BMP_HEADER_SIZE, file);
}

void get_color(double value, unsigned char *r, unsigned char *g, unsigned char *b) {
    if (value >= 500.0) { *r = 255; *g = 0; *b = 0; }
    else if (value >= 100.0) { *r = 255; *g = 128; *b = 0; }
    else if (value >= 50.0) { *r = 171; *g = 71; *b = 188; }
    else if (value >= 25.0) { *r = 255; *g = 255; *b = 0; }
    else if (value >= 1.0) { *r = 0; *g = 0; *b = 255; }
    else if (value >= 0.1) { *r = 5; *g = 248; *b = 252; }
    else { *r = 255; *g = 255; *b = 255; }
}

void write_grid(FILE *file, double *grid, int nx, int ny) {
    for (int i = nx - 1; i >= 0; i--) {
        for (int j = 0; j < ny; j++) {
            unsigned char r, g, b;
            get_color(grid[i * ny + j], &r, &g, &b);
            fwrite(&b, 1, 1, file);
            fwrite(&g, 1, 1, file);
            fwrite(&r, 1, 1, file);
        }
        for (int p = 0; p < (4 - (ny * 3) % 4) % 4; p++) fputc(0, file);
    }
}

int main(int argc, char *argv[]) {
    if (argc < 4 || argc > 6) {
        printf("Usage: %s <grid_size> <steps> <output.bmp> [BLOCK_X BLOCK_Y]\n", argv[0]);
        return 1;
    }

    int nx = atoi(argv[1]);
    int ny = nx;
    int steps = atoi(argv[2]);
    double r = ALPHA * DT / (DX * DY);

    int BLOCK_X = 16;
    int BLOCK_Y = 16;
    if (argc == 6) {
        BLOCK_X = atoi(argv[4]);
        BLOCK_Y = atoi(argv[5]);
    }

    size_t size = nx * ny * sizeof(double);
    double *h_grid = (double *)calloc(nx * ny, sizeof(double));
    double *h_result = (double *)calloc(nx * ny, sizeof(double));

    double *d_grid, *d_new_grid;
    hipMalloc((void **)&d_grid, size);
    hipMalloc((void **)&d_new_grid, size);

    initialize_grid(h_grid, nx, ny);
    hipMemcpy(d_grid, h_grid, size, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_X, BLOCK_Y);
    dim3 gridDim((ny + BLOCK_X - 1) / BLOCK_X, (nx + BLOCK_Y - 1) / BLOCK_Y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int t = 0; t < steps; t++) {
        heat_step<<<gridDim, blockDim>>>(d_grid, d_new_grid, nx, ny, r);
        double *tmp = d_grid;
        d_grid = d_new_grid;
        d_new_grid = tmp;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time: %.3f ms for grid %dx%d and %d steps\n", milliseconds, nx, ny, steps);

    hipMemcpy(h_result, d_grid, size, hipMemcpyDeviceToHost);

    FILE *f = fopen(argv[3], "wb");
    if (!f) {
        printf("Error opening output file.\n");
        return 1;
    }
    write_bmp_header(f, nx, ny);
    write_grid(f, h_result, nx, ny);
    fclose(f);

    free(h_grid); free(h_result);
    hipFree(d_grid); hipFree(d_new_grid);

    return 0;
}
